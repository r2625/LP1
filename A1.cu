#include "hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<math.h>

void init_array(float *a, const int N);
__global__
void sum(float* input)
{
 const int tid = threadIdx.x;
 int no_threads = blockDim.x;

 int step_size =1;

 while(no_threads>0)
 {
   if(tid<no_threads)
   {
      const int fst = tid*step_size*2;
      const int snd = fst + step_size;
      input[fst] += input[snd];
   }
   step_size <<=1;
   no_threads >>= 1;

 }
}

__global__
void min(float* input)
{
 const int tid = threadIdx.x;
 int no_threads = blockDim.x;

 int step_size =1;

 while(no_threads>0)
 {
   if(tid<no_threads)
   {
      const int fst = tid*step_size*2;
      const int snd = fst + step_size;
      if(input[snd]<input[fst])
         input[fst]  = input[snd];
   }
   step_size <<=1;
   no_threads >>= 1;

 }


}

__global__
void max(float* input)
{
 const int tid = threadIdx.x;
 int no_threads = blockDim.x;

 int step_size =1;

 while(no_threads>0)
 {
   if(tid<no_threads)
   {
      const int fst = tid*step_size*2;
      const int snd = fst + step_size;
      if(input[snd]>input[fst])
         input[fst]  = input[snd];
   }
   step_size <<=1;
   no_threads >>= 1;

 }


}


__global__
void std_(float* input,float avg)
{
 const int tid = threadIdx.x;
 int no_threads = blockDim.x;

 int step_size =1;

 while(no_threads>0)
 {
   if(tid<no_threads)
   {
      const int fst = tid*step_size*2;
      const int snd = fst + step_size;
      input[fst] = (input[fst]-avg)*(input[fst]-avg);
      input[snd] = (input[snd]-avg)*(input[snd]-avg);
      input[fst] += input[snd];
   }
   step_size <<=1;
   no_threads >>= 1;

 }


}


int main()
{

 srand(time(NULL));
 const int N = 4;
 const int size = N*sizeof(float);
 float *a;
 float *d_a,*d_b;
 float result, avg;
 double time_taken;
    
 a = (float*)malloc(sizeof(float)*N);
 //initialising the array
 init_array(a,N);

 //printing the array
 for(int i=0;i<N;i++)
   printf("%f   ",a[i]);

 hipMalloc(&d_a,size);
 hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);

 hipMalloc(&d_b,size);
 hipMemcpy(d_b,a,size,hipMemcpyHostToDevice);
 
 //----------------Sum--------------------------
 clock_t t;
 t = clock();
 sum<<<1,N/2>>>(d_a);
 hipMemcpy(&result,d_a,sizeof(float),hipMemcpyDeviceToHost);
 t = clock() - t;
 time_taken = ((double)t)/CLOCKS_PER_SEC; //in seconds
 printf(" Time taken by sum :%f",time_taken);
 printf("   Sum:  %f",result);
    	
 //----------------Min--------------------------
 t = clock();
 min<<<1,N/2>>>(d_a);
 hipMemcpy(&result,d_a,sizeof(float),hipMemcpyDeviceToHost);
 t = clock() - t;
 time_taken = ((double)t)/CLOCKS_PER_SEC; //in seconds
 printf(" Time taken by min :%f",time_taken);
 printf("   Min:  %f",result);
      
 //----------------Max--------------------------
 t = clock();
 max<<<1,N/2>>>(d_a); 
 t = clock() - t;
 time_taken = ((double)t)/CLOCKS_PER_SEC; //in seconds
 printf(" Time taken by max :%f",time_taken);
 hipMemcpy(&result,d_a,sizeof(float),hipMemcpyDeviceToHost);
 printf("   Max:  %f",result);
    
 //----------------Average--------------------
 t = clock();
 sum<<<1,N/2>>>(d_a);
 hipMemcpy(&result,d_a,sizeof(float),hipMemcpyDeviceToHost);
 avg = result/N;
 t = clock() - t;
 time_taken = ((double)t)/CLOCKS_PER_SEC; //in seconds
 printf(" Time taken by avg :%f",time_taken);
 printf("   Avg:  %f",avg);
 
    
 //----------------Standard deviation-------------
  t = clock();
 std_<<<1,N/2>>>(d_a,avg);
 float std;
 hipMemcpy(&std,d_b,sizeof(float),hipMemcpyDeviceToHost);
 std =std/N;
 std = sqrt(std);
 t = clock() - t;
 time_taken = ((double)t)/CLOCKS_PER_SEC; //in seconds
 printf(" Time taken by std :%f",time_taken);
 printf(" STD IS:%f",std);

 
 hipFree(d_a);
 hipFree(d_b);
 delete[] a;

 return 0;
}


void init_array(float*a,const int N)
{
  for(int i=0;i<N;i++)
     a[i] = rand()%N + 1;
}
